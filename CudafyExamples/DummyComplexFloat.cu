
#include <hip/hip_runtime.h>
struct DummyComplexFloat
{
	public: float Real;
	public: float Imag;

	__device__ DummyComplexFloat()
	{
		Real = 0;
		Imag = 0;
	}

	// Methods
	__device__ DummyComplexFloat(float  r, float  i)
	{
		Real = r;
		Imag = i;
	}


	__device__ DummyComplexFloat  Add(DummyComplexFloat  c)
	{
		return DummyComplexFloat((Real + c.Real), (Imag + c.Imag));
	}
};